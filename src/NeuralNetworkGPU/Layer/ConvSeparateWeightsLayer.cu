#include "hip/hip_runtime.h"
//
// Created by przemo on 27.12.2019.
//

#include "ConvSeparateWeightsLayer.h"

namespace NeuralNetworkGPU
{

	/*
	 *
	 */
	__global__ void determineOutputFuncConvSW(float *t_input, TensorSize *t_inputSize,
			float *t_output,
			float *t_sums,
			float *t_weights, MatrixSize *t_filterSize,
			float *t_deltas,
			float *d_b)
	{
		long index = blockIdx.x + blockIdx.y*gridDim.x + threadIdx.x*gridDim.x*gridDim.y;

		//sums x[i]*w[i]
		int yFrame = t_inputSize->x*t_inputSize->y;
		int yfFrame = t_filterSize->y*t_filterSize->x;
		int yOffset = blockIdx.y*t_inputSize->x;
		int zfOffset = yfFrame*t_inputSize->z*index;
		float sum = 0;

		for(int y=0,yf=0,yi=yOffset; y<t_filterSize->y; y++)
		{
			for(int x=0; x<t_filterSize->x; x++)
			{
				for(int z=0,zf=zfOffset,zi=0; z<t_inputSize->z; z++)
				{
					sum += t_input[blockIdx.x+x + yi + zi] * t_weights[x + yf + zf];
					zf+=yfFrame;
					zi+=yFrame;
				}
			}
			yf+=t_filterSize->x;
			yi+=t_inputSize->y;
		}

		t_sums[index] = sum;
//		sum = sum > 255 ? 255 : sum;
//		sum = sum < -255 ? -255 : sum;
		//reset delta
		t_deltas[index] = 0;
		//activation function
		t_output[index] =
				1 / (1 + exp(-(*d_b)*sum) );	//sigmoid function
//				sum > 0 ? sum : sum*0.05; 		//RELU function
	}

	/*
	 *
	 */
	__global__ void learnSGDConvSW(float *t_input, TensorSize *t_inputSize,
			float *t_output,
			float *t_sums,
			float *t_weights, MatrixSize *t_filterSize,
			float *t_deltas, float *t_prevDeltas,
			float *d_n,float *d_b)
	{
		long index = blockIdx.x + blockIdx.y*gridDim.x + threadIdx.x*gridDim.x*gridDim.y;
		float delta = t_deltas[index];

		//determine common multiplier
		float e = exp(-(*d_b)*t_sums[index]);
		float m = 1 + e;
		float derivative = ((*d_b)*e/(m*m));
//		float derivative = t_sums[index] > 0 ? 1 : 0.05;

		float p = (*d_n)* delta * derivative;
		//calculate new weights
		int yFrame = t_inputSize->x*t_inputSize->y;
		int yfFrame = t_filterSize->y*t_filterSize->x;
		int yOffset = blockIdx.y*t_inputSize->x;
		int zfOffset = yfFrame*t_inputSize->z*index;
		for(int y=0,yf=0,yi=yOffset; y<t_filterSize->y; y++)
		{
			for(int x=0; x<t_filterSize->x; x++)
			{
				for(int z=0,zf=zfOffset,zi=0; z<t_inputSize->z; z++)
				{
					t_weights[ x + yf + zf ] -= p*t_input[blockIdx.x+x + yi + zi];

					zf+=yfFrame;
					zi+=yFrame;
				}
			}
			yf+=t_filterSize->x;
			yi+=t_inputSize->y;
		}

		//set delta to deeper neurons
		if(t_prevDeltas != nullptr)
		{
			float dd = delta*derivative;
			for(int y=0,yf=0,yi=yOffset; y<t_filterSize->y; y++)
			{
				for(int x=0; x<t_filterSize->x; x++)
				{
					for(int z=0,zf=0,zi=0; z<t_inputSize->z; z++)
					{
						t_prevDeltas[blockIdx.x+x + yi + zi] += dd * t_weights[ x + yf + zf ];

						zf+=yfFrame;
						zi+=yFrame;
					}
				}
				yf+=t_filterSize->x;
				yi+=t_inputSize->y;
			}
		}

		//reset delta
		t_deltas[index] = 0;

	}

	/*
	 *
	 */
	__global__ void learnAdamConvSW(float *t_input, TensorSize *t_inputSize,
			float *t_output,
			float *t_sums,
			float *t_weights, MatrixSize *t_filterSize,
			float *t_deltas, float *t_prevDeltas,
			float *t_m,float *t_v,
			float *t_n,float *t_b,
			float *t_B1,float *t_B2)
	{
		long index = blockIdx.x + blockIdx.y*gridDim.x + threadIdx.x*gridDim.x*gridDim.y;
		float delta = t_deltas[index];

		//determine derivative and gradients
		float e = exp(-(*t_b)*t_sums[index]);
		float m = 1 + e;
		float derivative = ((*t_b)*e/(m*m));
//		float sum = t_sums[index];
//		float derivative = sum > 0 && sum < 65536 ? 1 : 0.05;
		float grad = delta*derivative; // gradient without x factor
		float grad2 = grad*grad;

		//calculate new weights
		int yFrame = t_inputSize->x*t_inputSize->y;
		int yfFrame = t_filterSize->y*t_filterSize->x;
		int yOffset = blockIdx.y*t_inputSize->x;
		int zfOffset = yfFrame*t_inputSize->z*index;
		float mTarget,vTarget;
		float mNew, vNew;
		for(int y=0,yf=0,yi=yOffset; y<t_filterSize->y; y++)
		{
			for(int x=0; x<t_filterSize->x; x++)
			{
				for(int z=0,zf=zfOffset,zi=0; z<t_inputSize->z; z++)
				{
					float input = t_input[blockIdx.x+x + yi + zi];
					//calculate new m & v
					mTarget = grad*input;
					vTarget = grad2*input*input;
					mNew = mTarget - (*t_B1)*(mTarget-t_m[x + yf + zf]);
					vNew = vTarget - (*t_B2)*(vTarget-t_v[x + yf + zf]);
					t_m[x + yf + zf] = mNew;
					t_v[x + yf + zf] = vNew;

					//update weights
					t_weights[x + yf + zf] -= __fdiv_rd ((*t_n)*mNew , (__fsqrt_rd(vNew)+0.0000001));

					zf+=yfFrame;
					zi+=yFrame;
				}
			}
			yf+=t_filterSize->x;
			yi+=t_inputSize->y;
		}

		//set delta to deeper neurons
		if(t_prevDeltas != nullptr)
		{
			float dd = delta*derivative;
			for(int y=0,yf=0,yi=yOffset; y<t_filterSize->y; y++)
			{
				for(int x=0; x<t_filterSize->x; x++)
				{
					for(int z=0,zf=zfOffset,zi=0; z<t_inputSize->z; z++)
					{
						t_prevDeltas[blockIdx.x+x + yi + zi] += dd * t_weights[ x + yf + zf ];

						zf+=yfFrame;
						zi+=yFrame;
					}
				}
				yf+=t_filterSize->x;
				yi+=t_inputSize->y;
			}
		}

		//reset delta
		t_deltas[index] = 0;

	}

	/*
	 *
	 */
	__global__ void scaleWeightsConvSW(TensorSize *t_inputSize,
			float *t_weights, MatrixSize *t_filterSize)
	{
		//calculate new weights
		int yfFrame = t_filterSize->y*t_filterSize->x;
		int zfOffset = yfFrame*t_inputSize->z*threadIdx.x;
		float sum = 0;
		for(int y=0,yf=0; y<t_filterSize->y; y++)
		{
			for(int x=0; x<t_filterSize->x; x++)
			{
				for(int z=0,zf=zfOffset; z<t_inputSize->z; z++)
				{
					sum = abs(t_weights[ x + yf + zf ]) > sum ? abs(t_weights[ x + yf + zf ]) : sum;

					zf+=yfFrame;
				}
			}
			yf+=t_filterSize->x;
		}
		__fdiv_rd(sum,100);
		for(int y=0,yf=0; y<t_filterSize->y; y++)
		{
			for(int x=0; x<t_filterSize->x; x++)
			{
				for(int z=0,zf=zfOffset; z<t_inputSize->z; z++)
				{
					t_weights[ x + yf + zf ] = __fdiv_rd(t_weights[ x + yf + zf ],sum);

					zf+=yfFrame;
				}
			}
			yf+=t_filterSize->x;
		}
	}

	/*
	 *
	 */
	ConvSeparateWeightsLayer::ConvSeparateWeightsLayer(float t_parameterB, float t_learnRate, int convLayers,
			MatrixSize t_filterSize, NeuronsPtr t_prevLayerReference)
	{
		float b1 = 0.9, b2 = 0.999;

		size = TensorSize(t_prevLayerReference.tSize.x-t_filterSize.x+1,
						  t_prevLayerReference.tSize.y-t_filterSize.y+1,
						  convLayers);
		de_input = t_prevLayerReference.inputPtr;

		//learn rate
		hipMalloc( (void **) &d_n, sizeof(float));
		hipMemcpy(d_n, &(t_learnRate), sizeof(float), hipMemcpyHostToDevice);
		//parameter b
		hipMalloc( (void **) &d_b, sizeof(float));
		hipMemcpy(d_b, &(t_parameterB), sizeof(float), hipMemcpyHostToDevice);
		//Adam parameters
		hipMalloc( (void **) &d_B1, sizeof(float));
		hipMemcpy(d_B1, &(b1), sizeof(float), hipMemcpyHostToDevice);
		hipMalloc( (void **) &d_B2, sizeof(float));
		hipMemcpy(d_B2, &(b2), sizeof(float), hipMemcpyHostToDevice);

		//input size
		hipMalloc( (void **) &d_inputSize, sizeof(TensorSize));
		hipMemcpy(d_inputSize, &t_prevLayerReference.tSize, sizeof(TensorSize), hipMemcpyHostToDevice);
		inputSize = t_prevLayerReference.tSize;

		//output
		hipMalloc( (void **) &d_output, sizeof(float)*size.m);
		output = (float*) std::malloc(sizeof(float)*size.m);

		//filter size
		filterSize = t_filterSize;
		hipMalloc( (void **) &d_filterSize, sizeof(MatrixSize));
		hipMemcpy(d_filterSize, &t_filterSize, sizeof(MatrixSize), hipMemcpyHostToDevice);
		//weights
		int weightsSize = t_filterSize.m*t_prevLayerReference.tSize.z*size.z*size.y*size.x;
		hipMalloc( (void **) &d_weights, sizeof(float)*weightsSize);
		initWeights();

		//sums
		hipMalloc( (void **) &d_sums, sizeof(float)*size.m);
		//deltas
		hipMalloc( (void **) &d_deltas, sizeof(float)*size.m);
		deltas = (float*) malloc(sizeof(float)*size.m);
		de_prevDeltas = t_prevLayerReference.deltaPtr;

		//adam learn
		float *zeros = (float*) malloc(sizeof(float)*weightsSize);
		for(int i=0; i<weightsSize; i++)	zeros[i] = 0;

		hipMalloc( (void **) &d_m, sizeof(float)*weightsSize);
		hipMemcpy(d_m, zeros, sizeof(float)*weightsSize, hipMemcpyHostToDevice);
		hipMalloc( (void **) &d_v, sizeof(float)*weightsSize);
		hipMemcpy(d_v, zeros, sizeof(float)*weightsSize, hipMemcpyHostToDevice);

		free(zeros);

	}

	/*
	 *
	 */
	ConvSeparateWeightsLayer::~ConvSeparateWeightsLayer()
	{
		hipFree(d_n);
		hipFree(d_b);
		hipFree(d_B1);
		hipFree(d_B2);

		hipFree(d_inputSize);
		hipFree(d_output);
		hipFree(d_sums);
		hipFree(d_weights);

		hipFree(d_filterSize);

		hipFree(d_deltas);

		hipFree(d_m);
		hipFree(d_v);

		free(output);
		free(deltas);
	}

	/*
	 *
	 */
	void ConvSeparateWeightsLayer::initWeights()
	{
		long weightsSize = filterSize.m*inputSize.z*size.z*size.y*size.x;

		float *randomValues = (float*) malloc(sizeof(float)*weightsSize);

		for(int i=0; i< weightsSize; i++)
		{
			float randomValue = getRandomWeight();
			randomValues[i] = randomValue;
		}
		hipMemcpy(d_weights, randomValues, sizeof(float)*weightsSize, hipMemcpyHostToDevice);
		free(randomValues);
	}

	/*
	 *
	 */
	std::vector<double> ConvSeparateWeightsLayer::getOutput()
	{
		hipMemcpy(output, d_output, sizeof(float)*size.m, hipMemcpyDeviceToHost);

		std::vector<double> result;
		int outputSize = size.multiply();
		for(int i=0; i<outputSize; i++ )
		{
			double v = output[i];
			result.push_back(v);
		}

		return result;
	}

	void ConvSeparateWeightsLayer::determineOutput()
	{
		dim3 threadsPerBlock(size.z);
		dim3 numBlocks(size.x, size.y);
		determineOutputFuncConvSW<<< numBlocks , threadsPerBlock >>>(de_input, d_inputSize,
																  d_output,
																  d_sums,
																  d_weights, d_filterSize,
																  d_deltas,
																  d_b);
	}

	void ConvSeparateWeightsLayer::learnSGD()
	{
//		int64 timeBefore = cv::getTickCount();
		dim3 threadsPerBlock(size.z);
		dim3 numBlocks(size.x, size.y);
		learnSGDConvSW<<< numBlocks , threadsPerBlock >>>(de_input, d_inputSize,
														d_output,
														d_sums,
														d_weights, d_filterSize,
														d_deltas, de_prevDeltas,
														d_n, d_b);
//		int64 afterBefore = cv::getTickCount();
//		std::cout << "Sigm: " << (afterBefore - timeBefore)/ cv::getTickFrequency() << "\n";
	}

	void ConvSeparateWeightsLayer::learnAdam()
	{
		dim3 threadsPerBlock(size.z);
		dim3 numBlocks(size.x, size.y);
		learnAdamConvSW<<< numBlocks , threadsPerBlock >>>(de_input, d_inputSize,
														d_output,
														d_sums,
														d_weights, d_filterSize,
														d_deltas, de_prevDeltas,
														d_m, d_v,
														d_n, d_b,
														d_B1, d_B2);
	}

	/*
	 *
	 */
	NeuronsPtr ConvSeparateWeightsLayer::getNeuronPtr()
	{
		return NeuronsPtr(layerId, d_output,size, d_deltas);
	}

	/*
	 *
	 */
	void ConvSeparateWeightsLayer::drawLayer()
	{
		std::vector<double> output = getOutput();
		for(int z=0; z<size.z; z++)
		{
			cv::Mat image = cv::Mat(size.y, size.x, CV_8UC3);
			for(int y=0; y<size.y; y++)
			{
				for(int x=0; x<size.x; x++)
				{
					uchar* ptrDst = image.ptr(y)+(x+x+x);
					int src = output[z*size.x*size.y + y*size.x + x]*255;
					ptrDst[0] = src;
					ptrDst[1] = src;
					ptrDst[2] = src;
				}
			}
			cv::resize(image, image, cv::Size(), 8, 8,CV_INTER_NN);
			//Print
			imshow(std::to_string(z), image);
			cv::waitKey(3);
		}
	}
}
