#include "hip/hip_runtime.h"
//
// Created by przemo on 27.12.2019.
//

#include "PoolingLayer.h"

namespace NeuralNetworkGPU
{

	/*
	 *
	 */
	__global__ void determineOutputFuncPool(float *t_input, TensorSize *t_inputSize,
			float *t_output,
			float *t_deltas)
	{

		long indexDst = threadIdx.x + threadIdx.y*blockDim.x + blockIdx.x*blockDim.y*blockDim.x;
		long indexSrc = threadIdx.x+threadIdx.x +
						(threadIdx.y+threadIdx.y)*(blockDim.x+blockDim.x) +
						blockIdx.x*(blockDim.y+blockDim.y)*(blockDim.x+blockDim.x);

		float value = 0;
		int maxXSize = t_inputSize->x;
		int maxYSize = t_inputSize->y;
		value += t_input[indexSrc];
		value += maxXSize > threadIdx.x ? t_input[indexSrc+1] : 0;
		value += maxYSize > threadIdx.y ? t_input[indexSrc+(threadIdx.y+threadIdx.y)*blockDim.x] : 0;
		value += maxYSize > threadIdx.y && maxXSize > threadIdx.x ? t_input[indexSrc+(threadIdx.y+threadIdx.y)*blockDim.x+1] : 0;
		value = __fdiv_rd(value,4);

		t_output[indexDst] = value;
		t_deltas[indexDst] = 0;
	}

	/*
	 *
	 */
	__global__ void learnFuncPool(float *t_input, TensorSize *t_inputSize,
			float *t_output,
			float *t_deltas, float *t_prevDeltas)
	{
		long indexDst = threadIdx.x + threadIdx.y*blockDim.x + blockIdx.x*blockDim.y*blockDim.x;
		long indexSrc = threadIdx.x+threadIdx.x +
						(threadIdx.y+threadIdx.y)*(blockDim.x+blockDim.x) +
						blockIdx.x*(blockDim.y+blockDim.y)*(blockDim.x+blockDim.x);

		float delta = __fdiv_rd(t_deltas[indexDst],4);

		//set delta to deeper neurons
		if(t_prevDeltas != nullptr)
		{
			int maxXSize = t_inputSize->x;
			int maxYSize = t_inputSize->y;
			t_prevDeltas[indexSrc] = delta;
			if(maxXSize > threadIdx.x) t_prevDeltas[indexSrc+1] = delta;
			if(maxYSize > threadIdx.y) t_prevDeltas[indexSrc+(threadIdx.y+threadIdx.y)*blockDim.x] = delta;
			if(maxXSize > threadIdx.x && maxYSize > threadIdx.y) t_prevDeltas[indexSrc+(threadIdx.y+threadIdx.y)*blockDim.x+1] = delta;
		}
		//reset delta
		t_deltas[indexDst] = 0;
	}

	/*
	 *
	 */
	PoolingLayer::PoolingLayer(NeuronsPtr t_prevLayerReference)
	{
		size = TensorSize((t_prevLayerReference.tSize.x+1)/2,(t_prevLayerReference.tSize.y+1)/2,t_prevLayerReference.tSize.z);
		de_input = t_prevLayerReference.inputPtr;

		//Input/output
		hipMalloc( (void **) &d_inputSize, sizeof(TensorSize));
		hipMemcpy(d_inputSize, &(t_prevLayerReference.tSize), sizeof(TensorSize), hipMemcpyHostToDevice);
		inputSize = t_prevLayerReference.tSize;

		hipMalloc( (void **) &d_output, sizeof(float)*size.m);
		output = (float*) std::malloc(sizeof(float)*size.m);

		//basic to learn
		hipMalloc( (void **) &d_deltas, sizeof(float)*size.m);
		deltas = (float*) malloc(sizeof(float)*size.m);
		de_prevDeltas = t_prevLayerReference.deltaPtr;
	}

	/*
	 *
	 */
	PoolingLayer::~PoolingLayer()
	{
		hipFree(d_inputSize);
		hipFree(d_output);

		hipFree(d_deltas);

		free(output);
		free(deltas);
	}

	/*
	 *
	 */
	std::vector<double> PoolingLayer::getOutput()
	{
		hipMemcpy(output, d_output, sizeof(float)*size.m, hipMemcpyDeviceToHost);

		std::vector<double> result;
		for(int i=0; i<size.m; i++ )
		{
			double v = output[i];
			result.push_back(v);
		}

		return result;
	}

	void PoolingLayer::determineOutput()
	{
		dim3 threadsPerBlock(size.x, size.y);
		dim3 numBlocks(size.z);
		determineOutputFuncPool<<< threadsPerBlock , numBlocks >>>(de_input, d_inputSize,
																	    d_output,
																	    d_deltas);
	}

	void PoolingLayer::learnSGD()
	{
//		int64 timeBefore = cv::getTickCount();
		dim3 threadsPerBlock(size.x, size.y);
		dim3 numBlocks(size.z);
		learnFuncPool<<< threadsPerBlock , numBlocks >>>(de_input, d_inputSize,
															 d_output,
															 d_deltas, de_prevDeltas);
//		int64 afterBefore = cv::getTickCount();
//		std::cout << "Sigm: " << (afterBefore - timeBefore)/ cv::getTickFrequency() << "\n";
	}

	void PoolingLayer::learnAdam()
	{
//		int64 timeBefore = cv::getTickCount();
		dim3 threadsPerBlock(size.x, size.y);
		dim3 numBlocks(size.z);
		learnFuncPool<<< threadsPerBlock , numBlocks >>>(de_input, d_inputSize,
															 d_output,
															 d_deltas, de_prevDeltas);
//		int64 afterBefore = cv::getTickCount();
//		std::cout << "Sigm: " << (afterBefore - timeBefore)/ cv::getTickFrequency() << "\n";
	}

	/*
	 *
	 */
	NeuronsPtr PoolingLayer::getNeuronPtr()
	{
		return NeuronsPtr(d_output,size, d_deltas);
	}
}
