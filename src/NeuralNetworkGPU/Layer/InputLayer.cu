//
// Created by przemo on 27.12.2019.
//

#include "InputLayer.h"

namespace NeuralNetworkGPU
{

	/*
	 *
	 */
	InputLayer::InputLayer(int t_size)
	{
//				int *d_a, *d_b;
//				int a=15;int b;
//				hipMalloc( (void **) &d_a, sizeof(int));
//				hipMalloc( (void **) &d_b, sizeof(int));
//
//				hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
//				funkcja<<<2,1>>>(d_a,d_b);
//				hipMemcpy(&b, d_b, sizeof(int), hipMemcpyDeviceToHost);
//
//				std::cout << a << "  " << b << "\n";
//
//				hipFree(d_a);
//				hipFree(d_b);
//
//				InputNeuron *d_n;
//				hipMalloc( (void **) &d_n, sizeof(InputNeuron));
//
//				double *d_in, *d_out;
//				double in=2.5;double out;
//				hipMalloc( (void **) &d_in, sizeof(double));
//				hipMalloc( (void **) &d_out, sizeof(double));
//
//				hipMemcpy(d_in, &in, sizeof(double), hipMemcpyHostToDevice);
//				funkcja2<<<1,1>>>(d_n,d_in,d_out);
//				hipMemcpy(&out, d_out, sizeof(double), hipMemcpyDeviceToHost);
//				std::cout << out << "\n";
//
//				hipFree(d_n);
//				hipFree(d_in);
//				hipFree(d_out);
		if(INPUT_BUFFER_SIZE < t_size)
		{
			std::cout << t_size;
			assert("CUDA input buffer to small");
		}
		size = t_size;
		input = (double*) malloc(sizeof(double)*size);
		hipMalloc( (void **) &d_input, sizeof(double)*size);

	}

	/*
	 *
	 */
	InputLayer::~InputLayer()
	{
		hipFree(d_input);
		free(input);
	}

	/*
	 *
	 */
	void InputLayer::setInput(std::vector<int> t_input)
	{
		assert(t_input.size() == size && "InputLayer::setInput input size not match");

		#pragma omp parallel for shared(input, t_input, size) private(i) default(none)
		for(int i=0; i<size; i++ )
		{
			input[i] = (double) t_input[i];
		}

		hipMemcpy(d_input, input, sizeof(double)*size, hipMemcpyHostToDevice);

	}

	/*
	 *
	 */
	void InputLayer::setInput(std::vector<double> t_input)
	{
		assert(t_input.size() == size && "InputLayer::setInput input size not match");

		for(int i=0; i<size; i++ )
		{
			input[i] = (double) t_input[i];
		}

		hipMemcpy(d_input, input, sizeof(double)*size, hipMemcpyHostToDevice);

	}

	/*
	 *
	 */
	std::vector<double> InputLayer::getOutput()
	{
		hipMemcpy(input, d_input, sizeof(double)*size, hipMemcpyDeviceToHost);

		std::vector<double> result;
		for(int i=0; i<size; i++ )
		{
			result.push_back(input[i]);
		}

		return result;
	}

	/*
	 *
	 */
	void InputLayer::determineOutput()
	{
		//Do nothing
//		funkcja3<<<1,size>>>(d_input);
	}

	/*
	 *
	 */
	void InputLayer::learnSGD()
	{
		//Do nothing
	}

	/*
	 *
	 */
	void InputLayer::learnAdam()
	{
		//Do nothing
	}

	/*
	 *
	 */
	NeuronsPtr InputLayer::getNeuronPtr()
	{
		return NeuronsPtr(d_input,size, nullptr);
	}

	/*
	 *
	 */
//	void InputLayer::saveToFile(std::ofstream &t_file)
//	{
//		t_file << (double) 0 << ' '; //Signature of InputLayer
//		t_file << (double) neurons.size() << ' ';
//	}
}
