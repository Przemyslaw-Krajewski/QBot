#include "hip/hip_runtime.h"
//
// Created by przemo on 27.12.2019.
//

#include "SigmoidLayer.h"

namespace NeuralNetworkGPU
{

	/*
	 *
	 */
	__global__ void determineOutputFunc(double *t_input, double *t_output, int *t_inputSize,
			double *t_sums,
			double *t_weights,
			double *t_deltas,
			double *d_b)
	{
		int inputSize = (*t_inputSize);
		//copy input to common buffer
		__shared__ double inputBuff[INPUT_BUFFER_SIZE];
		if(inputSize == blockDim.x)
		{
			inputBuff[threadIdx.x] = t_input[threadIdx.x];
		}
		else if(inputSize < blockDim.x)
		{
			if(threadIdx.x < inputSize) inputBuff[threadIdx.x] = t_input[threadIdx.x];
		}
		else if(inputSize > blockDim.x)
		{
			int index = inputSize-threadIdx.x-1;
			while(index >= 0)
			{
				inputBuff[index] = t_input[index];
				index -= blockDim.x;
			}
		}
		__syncthreads();

		//sums x[i]*w[i]
		long weightsIndex = inputSize*(threadIdx.x + blockIdx.x*blockDim.x);
		double sum = t_weights[weightsIndex];
		for(int i=0; i<inputSize;i++)
		{
			sum += inputBuff[i] * t_weights[ weightsIndex+i+1 ];
		}
		t_sums[threadIdx.x + blockIdx.x*blockDim.x] = sum;
		//activation function
		t_output[threadIdx.x + blockIdx.x*blockDim.x] = 1 / (1 + exp(-(*d_b)*sum) );
		//reset delta
		t_deltas[threadIdx.x + blockIdx.x*blockDim.x] = 0;
	}

	/*
	 *
	 */
	__global__ void learnSGDFunc(double *t_input, int *t_inputSize,
			double *t_output,
			double *t_sums,
			double *t_weights,
			double *t_deltas, double *t_prevDeltas,
			double *d_n,double *d_b)
	{
		int inputSize = *t_inputSize;

		//copy input to common buffer
		__shared__ double inputBuff[INPUT_BUFFER_SIZE];
		if(inputSize == blockDim.x)
		{
			inputBuff[threadIdx.x] = t_input[threadIdx.x];
		}
		else if(inputSize < blockDim.x)
		{
			if(threadIdx.x < inputSize) inputBuff[threadIdx.x] = t_input[threadIdx.x];
		}
		else if(inputSize > blockDim.x)
		{
			int index = inputSize-threadIdx.x-1;
			while(index >= 0)
			{
				inputBuff[index] = t_input[index];
				index -= blockDim.x;
			}
		}
		__syncthreads();

		long index = threadIdx.x +  blockIdx.x*blockDim.x;
		double delta = t_deltas[index];

		long weightsIndex = inputSize*(index);
		//determine common multiplier
		double e = exp(-(*d_b)*t_sums[index]);
		double m = 1 + e;
		double derivative = ((*d_b)*e/(m*m));

		double p = (*d_n)* delta * derivative;
		//calculate new weights
		//bias weight
		t_weights[weightsIndex] -= p;
		//rest weights
		for(int i=0; i<inputSize; i++)
		{
			t_weights[ weightsIndex+i+1 ] -= p*inputBuff[i];
		}

		//set delta to deeper neurons
		if(t_prevDeltas != nullptr)
		{
			for(int i=0; i<*t_inputSize; i++)
			{
				int idx = weightsIndex + i + 1;
				t_prevDeltas[i] += delta * derivative * t_weights[idx] ;
			}
		}

		//reset delta
		t_deltas[index] = 0;

	}

	/*
	 *
	 */
	__global__ void learnAdamFunc(double *t_input, int *t_inputSize,
			double *t_output,
			double *t_sums,
			double *t_weights,
			double *t_deltas, double *t_prevDeltas,
			double *t_m,double *t_v,
			double *t_n,double *t_b,
			double *t_B1,double *t_B2)
	{
		int inputSize = *t_inputSize;

		//copy input to common buffer
		__shared__ double inputBuff[INPUT_BUFFER_SIZE];
		if(inputSize == blockDim.x)
		{
			inputBuff[threadIdx.x] = t_input[threadIdx.x];
		}
		else if(inputSize < blockDim.x)
		{
			if(threadIdx.x < inputSize) inputBuff[threadIdx.x] = t_input[threadIdx.x];
		}
		else if(inputSize > blockDim.x)
		{
			int index = inputSize-threadIdx.x-1;
			while(index >= 0)
			{
				inputBuff[index] = t_input[index];
				index -= blockDim.x;
			}
		}
		__syncthreads();

		long index = threadIdx.x +  blockIdx.x*blockDim.x;
		double delta = t_deltas[index];

		long weightsIndex = inputSize*(index);
		//determine common multiplier
		double e = exp(-(*t_b)*t_sums[index]);
		double m = 1 + e;
		double derivative = ((*t_b)*e/(m*m));
		double grad = delta*derivative; // gradient without x factor
		double grad2 = grad*grad;

		//calculate moment vectors
		t_m[weightsIndex] = (*t_B1)*t_m[weightsIndex] + (1-(*t_B1))*grad;
		t_v[weightsIndex] = (*t_B2)*t_v[weightsIndex] + (1-(*t_B2))*grad*grad;
		for(int i=0; i<inputSize; i++)
		{
			t_m[weightsIndex+i+1] = (*t_B1)*t_m[weightsIndex+i+1] + (1-(*t_B1))*grad*inputBuff[i];
			t_v[weightsIndex+i+1] = (*t_B2)*t_v[weightsIndex+i+1] + (1-(*t_B2))*grad2*inputBuff[i]*inputBuff[i];
		}

		//calculate new weights
		t_weights[weightsIndex] -= (*t_n)*t_m[weightsIndex] / (__fsqrt_rd(t_v[weightsIndex]+0.00000001));
		for(int i=0; i<inputSize; i++)
		{
//			t_weights[ weightsIndex+i+1 ] -= p*inputBuff[i];
			t_weights[weightsIndex+i+1] -= (*t_n)*t_m[weightsIndex+i+1] / (__fsqrt_rd(t_v[weightsIndex+i+1]+0.00000001));
		}

		//set delta to deeper neurons
		if(t_prevDeltas != nullptr)
		{
			for(int i=0; i<*t_inputSize; i++)
			{
				int idx = weightsIndex + i + 1;
				t_prevDeltas[i] += grad * t_weights[idx] ;
			}
		}

		//reset delta
		t_deltas[index] = 0;

	}

	double SigmoidLayer::b = 0;

	/*
	 *
	 */
	SigmoidLayer::SigmoidLayer(double t_parameterB, double t_learnRate, int t_size, NeuronsPtr t_prevLayerReference)
	{
		double b1 = 0.9, b2 = 0.999;

		size = t_size;
		de_input = t_prevLayerReference.inputPtr;

		//Parameters
		hipMalloc( (void **) &d_n, sizeof(double));
		hipMemcpy(d_n, &(t_learnRate), sizeof(double), hipMemcpyHostToDevice);
		learnRate = t_learnRate;
		hipMalloc( (void **) &d_b, sizeof(double));
		hipMemcpy(d_b, &(t_parameterB), sizeof(double), hipMemcpyHostToDevice);
		hipMalloc( (void **) &d_B1, sizeof(double));
		hipMemcpy(d_B1, &(b1), sizeof(double), hipMemcpyHostToDevice);
		hipMalloc( (void **) &d_B2, sizeof(double));
		hipMemcpy(d_B2, &(b2), sizeof(double), hipMemcpyHostToDevice);

		//Input/output
		hipMalloc( (void **) &d_inputSize, sizeof(int));
		hipMemcpy(d_inputSize, &(t_prevLayerReference.size), sizeof(int), hipMemcpyHostToDevice);
		inputSize = t_prevLayerReference.size;

		hipMalloc( (void **) &d_output, sizeof(double)*size);
		output = (double*) std::malloc(sizeof(double)*size);

		//basic to learn
		hipMalloc( (void **) &d_sums, sizeof(double)*size);

		hipMalloc( (void **) &d_weights, sizeof(double)*size*(inputSize+1));
		initWeights();

		hipMalloc( (void **) &d_deltas, sizeof(double)*size);
		deltas = (double*) malloc(sizeof(double)*size);
		de_prevDeltas = t_prevLayerReference.deltaPtr;

		//additional to learn
		double *zeros = (double*) malloc(sizeof(double)*size*(inputSize+1));
		for(int i=0; i<(inputSize+1)*size; i++)	zeros[i] = 0;

		hipMalloc( (void **) &d_m, sizeof(double)*size*(inputSize+1));
		hipMemcpy(d_m, zeros, sizeof(double)*size*(inputSize+1), hipMemcpyHostToDevice);
		hipMalloc( (void **) &d_v, sizeof(double)*size*(inputSize+1));
		hipMemcpy(d_v, zeros, sizeof(double)*size*(inputSize+1), hipMemcpyHostToDevice);

		free(zeros);

		// split to blocks
		numberOfBlocks = 1;
		while(1)
		{
			numberOfThreads = size/numberOfBlocks;
			if(numberOfThreads<=800 && numberOfThreads*numberOfBlocks==size) break;
			numberOfBlocks++;

			assert(numberOfBlocks < 10 && "Could not match thread/block size");
		}

	}

	/*
	 *
	 */
	SigmoidLayer::~SigmoidLayer()
	{
		hipFree(d_n);
		hipFree(d_b);
		hipFree(d_B1);
		hipFree(d_B2);

		hipFree(d_inputSize);
		hipFree(d_output);

		hipFree(d_sums);
		hipFree(d_weights);
		hipFree(d_deltas);

		hipFree(d_m);
		hipFree(d_v);

		free(output);
		free(deltas);
	}

	/*
	 *
	 */
	void SigmoidLayer::initWeights()
	{
		double *randomValues = (double*) malloc(sizeof(double)*size*(inputSize+1));

		for(int i=0; i<(inputSize+1)*size; i++)
		{
			double randomValue = getRandomWeight();
			randomValues[i] = randomValue;

		}
		hipMemcpy(d_weights, randomValues, sizeof(double)*size*(inputSize+1), hipMemcpyHostToDevice);
		free(randomValues);
	}

	/*
	 *
	 */
	std::vector<double> SigmoidLayer::getOutput()
	{
		hipMemcpy(output, d_output, sizeof(double)*size, hipMemcpyDeviceToHost);

		std::vector<double> result;
		for(int i=0; i<size; i++ )
		{
			double v = output[i];
			result.push_back(v);
		}

		return result;
	}

	void SigmoidLayer::determineOutput()
	{
		determineOutputFunc<<< numberOfThreads , numberOfBlocks >>>(de_input, d_output, d_inputSize, d_sums, d_weights, d_deltas, d_b);
	}

	void SigmoidLayer::setDelta(std::vector<double> t_z)
	{
		assert(t_z.size() == size && "learning values size not match");

		#pragma omp parallel for shared(deltas,size,output, t_z) private(i) default(none)
		for(int i=0; i<size; i++ )
		{
			deltas[i] = (double) output[i] - t_z[i];
		}

		hipMemcpy(d_deltas, deltas, sizeof(double)*size, hipMemcpyHostToDevice);
	}

	void SigmoidLayer::learnSGD()
	{
//		int64 timeBefore = cv::getTickCount();
		learnSGDFunc<<< numberOfThreads , numberOfBlocks >>>(de_input, d_inputSize,
															 d_output,
															 d_sums,
															 d_weights,
															 d_deltas, de_prevDeltas,
															 d_n, d_b);
//		int64 afterBefore = cv::getTickCount();
//		std::cout << "Sigm: " << (afterBefore - timeBefore)/ cv::getTickFrequency() << "\n";
	}

	void SigmoidLayer::learnAdam()
	{
//		int64 timeBefore = cv::getTickCount();
		learnAdamFunc<<< numberOfThreads , numberOfBlocks >>>(de_input, d_inputSize,
															  d_output,
															  d_sums,
															  d_weights,
															  d_deltas, de_prevDeltas,
															  d_m, d_v,
															  d_n, d_b,
															  d_B1, d_B2);
//		int64 afterBefore = cv::getTickCount();
//		std::cout << "Sigm: " << (afterBefore - timeBefore)/ cv::getTickFrequency() << "\n";
	}

	/*
	 *
	 */
	NeuronsPtr SigmoidLayer::getNeuronPtr()
	{
		return NeuronsPtr(d_output,size, d_deltas);
	}

	/*
	 *
	 */
//	void SigmoidLayer::saveToFile(std::ofstream & t_file)
//	{
//		t_file << (double) 1 << ' '; //Signature of SigmoidLayer
//		t_file << (double) neurons.size() << ' ';
//		t_file << learnRate << ' ';
//		t_file << b << ' ';
//
//		for( auto it = neurons.begin(); it != neurons.end(); it++)
//		{
//			std::vector<double> *weights = it->getWeights();
//			for(int i=0; i<weights->size(); i++)
//			{
//				t_file << (*weights)[i] << ' ';
//			}
//		}
//	}

	/*
	 *
	 */
//	void SigmoidLayer::loadFromFile(std::ifstream & t_file)
//	{
//		for( auto it = neurons.begin(); it != neurons.end(); it++)
//		{
//			double buff;
//			std::vector<double> *weights = it->getWeights();
//			for(int i=0; i<weights->size(); i++)
//			{
//				if(t_file.eof()) {assert("SigmoidLayer LoadFromFile: unexpected end of file");}
//				t_file >> buff;
//				(*weights)[i] = buff;
//			}
//		}
//	}
}
