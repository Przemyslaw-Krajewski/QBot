#include "hip/hip_runtime.h"
//
// Created by przemo on 27.12.2019.
//

#include "SigmoidLayer.h"

#include "ActivationFunctions.h"

namespace NeuralNetworkGPU
{
	/*
	 *
	 */
	template<ActivationFunction F>
	__global__
	void determineOutputFunc(float *t_input, float *t_output, int *t_inputSize,
			float *t_sums,
			float *t_weights,
			float *t_deltas,
			float *d_b)
	{
		int inputSize = (*t_inputSize);
		//copy input to common buffer
		__shared__ float inputBuff[INPUT_BUFFER_SIZE];
		if(inputSize == blockDim.x)
		{
			inputBuff[threadIdx.x] = t_input[threadIdx.x];
		}
		else if(inputSize < blockDim.x)
		{
			if(threadIdx.x < inputSize) inputBuff[threadIdx.x] = t_input[threadIdx.x];
		}
		else if(inputSize > blockDim.x)
		{
			int index = inputSize-threadIdx.x-1;
			while(index >= 0)
			{
				inputBuff[index] = t_input[index];
				index -= blockDim.x;
			}
		}
		__syncthreads();

		//sums x[i]*w[i]
		long weightsIndex = inputSize*(threadIdx.x + blockIdx.x*blockDim.x);
		float sum = t_weights[weightsIndex];
		for(int i=0; i<inputSize;i++)
		{
			sum += inputBuff[i] * t_weights[ weightsIndex+i+1 ];
		}
		t_sums[threadIdx.x + blockIdx.x*blockDim.x] = sum;
		//activation function
		t_output[threadIdx.x + blockIdx.x*blockDim.x] = activationFunctionKernel<F>(&sum, d_b);
		//reset delta
		t_deltas[threadIdx.x + blockIdx.x*blockDim.x] = 0;
	}

	/*
	 *
	 */
	template<ActivationFunction F>
	__global__
	void learnSGDFunc(float *t_input, int *t_inputSize,
			float *t_output,
			float *t_sums,
			float *t_weights,
			float *t_deltas, float *t_prevDeltas,
			float *d_n,float *d_b)
	{
		int inputSize = *t_inputSize;

		//copy input to common buffer
		__shared__ float inputBuff[INPUT_BUFFER_SIZE];
		if(inputSize == blockDim.x)
		{
			inputBuff[threadIdx.x] = t_input[threadIdx.x];
		}
		else if(inputSize < blockDim.x)
		{
			if(threadIdx.x < inputSize) inputBuff[threadIdx.x] = t_input[threadIdx.x];
		}
		else if(inputSize > blockDim.x)
		{
			int index = inputSize-threadIdx.x-1;
			while(index >= 0)
			{
				inputBuff[index] = t_input[index];
				index -= blockDim.x;
			}
		}
		__syncthreads();

		long index = threadIdx.x +  blockIdx.x*blockDim.x;
		float delta = t_deltas[index];

		long weightsIndex = inputSize*(index);
		//determine common multiplier
		float derivative = derivativeFunctionKernel<F>(&t_sums[index],d_b);

		float p = (*d_n)* delta * derivative;
		//calculate new weights
		//bias weight
		t_weights[weightsIndex] -= p;
		//rest weights
		for(int i=0; i<inputSize; i++)
		{
			t_weights[ weightsIndex+i+1 ] -= p*inputBuff[i];
		}

		//set delta to deeper neurons
		if(t_prevDeltas != nullptr)
		{
			for(int i=0; i<*t_inputSize; i++)
			{
				int idx = weightsIndex + i + 1;
				t_prevDeltas[i] += delta * derivative * t_weights[idx] ;
			}
		}

		//reset delta
		t_deltas[index] = 0;

	}

	/*
	 *
	 */
	template<ActivationFunction F>
	__global__
	void learnAdamFunc(float *t_input, int *t_inputSize,
			float *t_output,
			float *t_sums,
			float *t_weights,
			float *t_deltas, float *t_prevDeltas,
			float *t_m,float *t_v,
			float *t_n,float *t_b,
			float *t_B1,float *t_B2)
	{
		int inputSize = *t_inputSize;

		//copy input to common buffer
		__shared__ float inputBuff[INPUT_BUFFER_SIZE];
		if(inputSize == blockDim.x)
		{
			inputBuff[threadIdx.x] = t_input[threadIdx.x];
		}
		else if(inputSize < blockDim.x)
		{
			if(threadIdx.x < inputSize) inputBuff[threadIdx.x] = t_input[threadIdx.x];
		}
		else if(inputSize > blockDim.x)
		{
			int index = inputSize-threadIdx.x-1;
			while(index >= 0)
			{
				inputBuff[index] = t_input[index];
				index -= blockDim.x;
			}
		}
		__syncthreads();

		long index = threadIdx.x +  blockIdx.x*blockDim.x;

		long weightsIndex = inputSize*(index);
		//determine common multiplier
		float derivative = derivativeFunctionKernel<F>(&t_sums[index],t_b);
		float grad = t_deltas[index]*derivative; // gradient without x factor
		float grad2 = grad*grad;

		//calculate new moment vectors and weights
		float mNew,vNew;
		mNew = grad - (*t_B1)*(grad-t_m[weightsIndex]);
		vNew = grad2 - (*t_B2)*(grad2-t_v[weightsIndex]);
		t_weights[weightsIndex] -= __fdiv_rd( (*t_n)*mNew , (__fsqrt_rd(vNew)+0.00001));
		t_m[weightsIndex] = mNew;
		t_v[weightsIndex] = vNew;

		float mTarget,vTarget;
		for(int i=0, indx=weightsIndex+1; i<inputSize; i++,indx++)
		{
			mTarget = grad*inputBuff[i];
			vTarget = grad2*inputBuff[i]*inputBuff[i];
			mNew = mTarget - (*t_B1)*(mTarget-t_m[indx]);
			vNew = vTarget - (*t_B2)*(vTarget-t_v[indx]);
			t_weights[indx] -= __fdiv_rd ((*t_n)*mNew , (__fsqrt_rd(vNew)+0.00001));
			t_m[indx] = mNew;
			t_v[indx] = vNew;
		}

		//set delta to deeper neurons
		if(t_prevDeltas != nullptr)
		{
			for(int i=0; i<*t_inputSize; i++)
			{
				t_prevDeltas[i] += grad * t_weights[weightsIndex+i+1] ;
			}
		}

		//reset delta
		t_deltas[index] = 0;

	}


	template class NeuralNetworkGPU::SigmoidLayer<NeuralNetworkGPU::ActivationFunction::Sigmoid>;
	template class NeuralNetworkGPU::SigmoidLayer<NeuralNetworkGPU::ActivationFunction::Linear>;
	template class NeuralNetworkGPU::SigmoidLayer<NeuralNetworkGPU::ActivationFunction::RELU>;
	template class NeuralNetworkGPU::SigmoidLayer<NeuralNetworkGPU::ActivationFunction::LeakRELU>;

	/*
	 *
	 */
	template<ActivationFunction F>
	SigmoidLayer<F>::SigmoidLayer(float t_parameterB, float t_learnRate, int t_size, NeuronsPtr t_prevLayerReference)
	{
		float b1 = 0.9, b2 = 0.999;

		prevLayerId = t_prevLayerReference.id;

		size = t_size;
		de_input = t_prevLayerReference.inputPtr;

		//Parameters
		hipMalloc( (void **) &d_n, sizeof(float));
		hipMemcpy(d_n, &(t_learnRate), sizeof(float), hipMemcpyHostToDevice);
		learnRate = t_learnRate;
		hipMalloc( (void **) &d_b, sizeof(float));
		hipMemcpy(d_b, &(t_parameterB), sizeof(float), hipMemcpyHostToDevice);
		hipMalloc( (void **) &d_B1, sizeof(float));
		hipMemcpy(d_B1, &(b1), sizeof(float), hipMemcpyHostToDevice);
		hipMalloc( (void **) &d_B2, sizeof(float));
		hipMemcpy(d_B2, &(b2), sizeof(float), hipMemcpyHostToDevice);

		//Input/output
		hipMalloc( (void **) &d_inputSize, sizeof(int));
		hipMemcpy(d_inputSize, &(t_prevLayerReference.size), sizeof(int), hipMemcpyHostToDevice);
		inputSize = t_prevLayerReference.size;

		hipMalloc( (void **) &d_output, sizeof(float)*size);
		output = (float*) std::malloc(sizeof(float)*size);

		//basic to learn
		hipMalloc( (void **) &d_sums, sizeof(float)*size);

		hipMalloc( (void **) &d_weights, sizeof(float)*size*(inputSize+1));
		initWeights();

		hipMalloc( (void **) &d_deltas, sizeof(float)*size);
		deltas = (float*) malloc(sizeof(float)*size);
		de_prevDeltas = t_prevLayerReference.deltaPtr;

		//additional to learn
		float *zeros = (float*) malloc(sizeof(float)*size*(inputSize+1));
		for(int i=0; i<(inputSize+1)*size; i++)	zeros[i] = 0;

		hipMalloc( (void **) &d_m, sizeof(float)*size*(inputSize+1));
		hipMemcpy(d_m, zeros, sizeof(float)*size*(inputSize+1), hipMemcpyHostToDevice);
		hipMalloc( (void **) &d_v, sizeof(float)*size*(inputSize+1));
		hipMemcpy(d_v, zeros, sizeof(float)*size*(inputSize+1), hipMemcpyHostToDevice);

		free(zeros);

		// split to blocks
		numberOfBlocks = 1;
		while(1)
		{
			numberOfThreads = size/numberOfBlocks;
			if(numberOfThreads<=800 && numberOfThreads*numberOfBlocks==size) break;
			numberOfBlocks++;

			assert(numberOfBlocks < 20 && "Could not match thread/block size");
		}
	}

	/*
	 *
	 */
	template<ActivationFunction F>
	SigmoidLayer<F>::~SigmoidLayer()
	{
		hipFree(d_n);
		hipFree(d_b);
		hipFree(d_B1);
		hipFree(d_B2);

		hipFree(d_inputSize);
		hipFree(d_output);

		hipFree(d_sums);
		hipFree(d_weights);
		hipFree(d_deltas);

		hipFree(d_m);
		hipFree(d_v);

		free(output);
		free(deltas);
	}

	/*
	 *
	 */
	template<ActivationFunction F>
	void SigmoidLayer<F>::initWeights()
	{
		float *randomValues = (float*) malloc(sizeof(float)*size*(inputSize+1));

		for(int i=0; i<(inputSize+1)*size; i++)
		{
			float randomValue = getRandomWeight();
			randomValues[i] = randomValue;

		}
		hipMemcpy(d_weights, randomValues, sizeof(float)*size*(inputSize+1), hipMemcpyHostToDevice);
		free(randomValues);
	}

	/*
	 *
	 */
	template<ActivationFunction F>
	void SigmoidLayer<F>::setWeights(float* t_weights)
	{
		hipMemcpy(d_weights, t_weights, sizeof(float)*size*(inputSize+1), hipMemcpyHostToDevice);
	}

	/*
	 *
	 */
	template<ActivationFunction F>
	void SigmoidLayer<F>::setMomentum1(float* t_momentum)
	{
		hipMemcpy(d_m, t_momentum, sizeof(float)*size*(inputSize+1), hipMemcpyHostToDevice);
	}

	/*
	 *
	 */
	template<ActivationFunction F>
	void SigmoidLayer<F>::setMomentum2(float* t_momentum)
	{
		hipMemcpy(d_v, t_momentum, sizeof(float)*size*(inputSize+1), hipMemcpyHostToDevice);
	}

	/*
	 *
	 */
	template<ActivationFunction F>
	std::vector<double> SigmoidLayer<F>::getOutput()
	{
		hipMemcpy(output, d_output, sizeof(float)*size, hipMemcpyDeviceToHost);

		std::vector<double> result;
		for(int i=0; i<size; i++ )
		{
			double v = output[i];
			result.push_back(v);
		}

		return result;
	}

	/*
	 *
	 */
	template<ActivationFunction F>
	void SigmoidLayer<F>::determineOutput()
	{
		determineOutputFunc<F><<< numberOfThreads , numberOfBlocks >>>(de_input, d_output, d_inputSize, d_sums, d_weights, d_deltas, d_b);
	}

	/*
	 *
	 */
	template<ActivationFunction F>
	void SigmoidLayer<F>::setDelta(std::vector<double> t_z)
	{
//		#pragma omp parallel for shared(deltas,size,output, t_z) private(i) default(none)
		for(int i=0; i<size; i++ )
		{
			deltas[i] = (float) t_z[i];
		}

		hipMemcpy(d_deltas, deltas, sizeof(float)*size, hipMemcpyHostToDevice);
	}

	/*
	 *
	 */
	template<ActivationFunction F>
	void SigmoidLayer<F>::learnSGD()
	{
//		int64 timeBefore = cv::getTickCount();
		learnSGDFunc<F><<< numberOfThreads , numberOfBlocks >>>(de_input, d_inputSize,
																						  d_output,
																						  d_sums,
																						  d_weights,
																						  d_deltas, de_prevDeltas,
																						  d_n, d_b);
//		int64 afterBefore = cv::getTickCount();
//		std::cout << "Sigm: " << (afterBefore - timeBefore)/ cv::getTickFrequency() << "\n";
	}

	/*
	 *
	 */
	template<ActivationFunction F>
	void SigmoidLayer<F>::learnAdam()
	{
//		int64 timeBefore = cv::getTickCount();
		learnAdamFunc<F><<< numberOfThreads , numberOfBlocks >>>(de_input,
																						   d_inputSize,
																						   d_output,
																						   d_sums,
																						   d_weights,
																						   d_deltas, de_prevDeltas,
																						   d_m, d_v,
																						   d_n, d_b,
																						   d_B1, d_B2);
//		int64 afterBefore = cv::getTickCount();
//		std::cout << "Sigm: " << (afterBefore - timeBefore)/ cv::getTickFrequency() << "\n";
	}

	/*
	 *
	 */
	template<ActivationFunction F>
	NeuronsPtr SigmoidLayer<F>::getNeuronPtr()
	{
		return NeuronsPtr(layerId, d_output,size, d_deltas);
	}

	/*
	 *
	 */
	template<ActivationFunction F>
	void SigmoidLayer<F>::saveToFile(std::ofstream & t_file)
	{
		t_file << (float) getLayerTypeId() << ' '; //Signature of SigmoidLayer
		t_file << (float) prevLayerId << ' '; 	   //Id of previous layer
		t_file << (float) size << ' ';
		t_file << (float) inputSize << ' ';
		t_file << (float) learnRate << ' ';
		float b;
		hipMemcpy(&b, d_b, sizeof(float), hipMemcpyDeviceToHost);
		t_file << b << ' ';

		float *weights = (float*) malloc(sizeof(float)*size*(inputSize+1));

		hipMemcpy(weights, d_weights, sizeof(float)*size*(inputSize+1), hipMemcpyDeviceToHost);
		for(int i=0; i<(inputSize+1)*size; i++)
		{
			t_file << weights[i] << ' ';
		}

		hipMemcpy(weights, d_m, sizeof(float)*size*(inputSize+1), hipMemcpyDeviceToHost);
		for(int i=0; i<(inputSize+1)*size; i++)
		{
			t_file << weights[i] << ' ';
		}

		hipMemcpy(weights, d_v, sizeof(float)*size*(inputSize+1), hipMemcpyDeviceToHost);
		for(int i=0; i<(inputSize+1)*size; i++)
		{
			t_file << weights[i] << ' ';
		}

		free(weights);
	}

	/*
	 *
	 */
	template<ActivationFunction F>
	SigmoidLayer<F>* SigmoidLayer<F>::loadFromFile(std::ifstream & t_file, std::vector<NeuronsPtr> &t_prevLayerReferences)
	{
		float size, inputSize, learnRate, b;
		float prevId;
		t_file >> prevId;
		t_file >> size;
		t_file >> inputSize;
		t_file >> learnRate;
		t_file >> b;

		SigmoidLayer<F>* layer = new SigmoidLayer<F>(b,learnRate,size,t_prevLayerReferences[(int)prevId]);

		float *weights = (float*) malloc(sizeof(float)*size*(inputSize+1));
		float buff;
		for(int i=0; i<(inputSize+1)*size; i++)
		{
			t_file >> buff;
			weights[i] = buff;
		}
		layer->setWeights(weights);

		for(int i=0; i<(inputSize+1)*size; i++)
		{
			t_file >> buff;
			weights[i] = buff;
		}
		layer->setMomentum1(weights);

		for(int i=0; i<(inputSize+1)*size; i++)
		{
			t_file >> buff;
			weights[i] = buff;
		}
		layer->setMomentum2(weights);

		free(weights);

		return layer;
	}

	/*
	 *
	 */
	template<ActivationFunction F>
	void SigmoidLayer<F>::drawLayer()
	{
		int blockSize=20;
		int spaceSize=2;
		std::vector<double> output = getOutput();

		cv::Mat mat = cv::Mat(1500, 1500, CV_8UC3);

		for(int x=0; x<mat.cols; x++)
		{
			for(int y=0; y<mat.rows; y++)
			{
				uchar* ptr = mat.ptr(y)+(x)*3;
				ptr[0] = 0;
				ptr[1] = 0;
				ptr[2] = 0;
			}
		}

		int x=spaceSize,y=spaceSize;
		for(double i : output)
		{
			for(int xx=0; xx<blockSize; xx++)
			{
				for(int yy=0; yy<blockSize; yy++)
				{
					uchar* ptr = mat.ptr(y+yy)+(x+xx)*3;
					ptr[2] = i < 0.5 ? 255-i*512 : 0;
					ptr[1] = i < 0.5 ? i*512 : 255-(i-0.5)*512;
					ptr[0] = i < 0.5 ? 0 : (i-0.5)*512;
					if(xx == 0 || yy == 0 || xx == blockSize-1 || yy == blockSize-1) {ptr[0] = ptr[1] = ptr[2] = 255;}
				}
			}
			x+=blockSize+spaceSize;
			if(x>mat.cols-blockSize-spaceSize)
			{
				y+=blockSize+spaceSize;
				x=spaceSize;
			}
		}

		//Print
		imshow("SigmoidLayer", mat);
		cv::waitKey(10);
	}

	/*
	 *
	 */
	template<ActivationFunction F>
	void SigmoidLayer<F>::printInfo()
	{
		std::cout << "	(" << layerId << ") Sigmoid <-- " << prevLayerId << " : ";
		std::cout << inputSize << " -> " << size << "   w:" << size*(inputSize+1) << "\n";
	}
}
