#include "hip/hip_runtime.h"
//
// Created by przemo on 27.12.2019.
//

#include "SigmoidLayer.h"

namespace NeuralNetworkGPU
{

	/*
	 *
	 */
	__global__ void determineOutputFunc(double *t_input, double *t_output, int *t_inputSize,
			double *t_sums,
			double *t_weights,
			double *t_deltas,
			double *d_b)
	{
		int inputSize = (*t_inputSize);
		//copy input to common buffer
		__shared__ double inputBuff[INPUT_BUFFER_SIZE];
		if(inputSize == blockDim.x)
		{
			inputBuff[threadIdx.x] = t_input[threadIdx.x];
		}
		else if(inputSize < blockDim.x)
		{
			if(threadIdx.x < inputSize) inputBuff[threadIdx.x] = t_input[threadIdx.x];
		}
		else if(inputSize > blockDim.x)
		{
			int index = inputSize-threadIdx.x-1;
			while(index >= 0)
			{
				inputBuff[index] = t_input[index];
				index -= blockDim.x;
			}
		}
		__syncthreads();

		//sums x[i]*w[i]
		long weightsIndex = inputSize*(threadIdx.x + blockIdx.x*blockDim.x);
		double sum = t_weights[weightsIndex];
		for(int i=0; i<inputSize;i++)
		{
			sum += inputBuff[i] * t_weights[ weightsIndex+i+1 ];
		}
		t_sums[threadIdx.x + blockIdx.x*blockDim.x] = sum;
		//activation function
		t_output[threadIdx.x + blockIdx.x*blockDim.x] = 1 / (1 + exp(-(*d_b)*sum) );
		//reset delta
		t_deltas[threadIdx.x + blockIdx.x*blockDim.x] = 0;
	}

	/*
	 *
	 */
	__global__ void learnBackPropagationFunc(double *t_input, int *t_inputSize,
			double *t_output,
			double *t_sums,
			double *t_weights,
			double *t_deltas, double *t_prevDeltas,
			double *d_n,double *d_b)
	{
		int inputSize = *t_inputSize;

		//copy input to common buffer
		__shared__ double inputBuff[INPUT_BUFFER_SIZE];
		if(inputSize == blockDim.x)
		{
			inputBuff[threadIdx.x] = t_input[threadIdx.x];
		}
		else if(inputSize < blockDim.x)
		{
			if(threadIdx.x < inputSize) inputBuff[threadIdx.x] = t_input[threadIdx.x];
		}
		else if(inputSize > blockDim.x)
		{
			int index = inputSize-threadIdx.x-1;
			while(index >= 0)
			{
				inputBuff[index] = t_input[index];
				index -= blockDim.x;
			}
		}
		__syncthreads();

		long index = threadIdx.x +  blockIdx.x*blockDim.x;
		double delta = t_deltas[index];

		long weightsIndex = inputSize*(index);
		//determine common multiplier
		double e = exp(-(*d_b)*t_sums[index]);
		double m = 1 + e;
		double derivative = ((*d_b)*e/(m*m));

		double p = (*d_n)* delta * derivative;
		//calculate new weights
		//bias weight
		t_weights[weightsIndex] -= p;
		//rest weights
		for(int i=0; i<inputSize; i++)
		{
			t_weights[ weightsIndex+i+1 ] -= p*inputBuff[i];
		}

		//set delta to deeper neurons
		if(t_prevDeltas != nullptr)
		{
			for(int i=0; i<*t_inputSize; i++)
			{
				int idx = weightsIndex + i + 1;
				t_prevDeltas[i] += delta * derivative * t_weights[idx] ;
			}
		}

		//reset delta
		t_deltas[index] = 0;

	}

	double SigmoidLayer::b = 0;

	/*
	 *
	 */
	SigmoidLayer::SigmoidLayer(double t_parameterB, double t_learnRate, int t_size, NeuronsPtr t_prevLayerReference)
	{
		size = t_size;
		de_input = t_prevLayerReference.inputPtr;

		hipMalloc( (void **) &d_n, sizeof(double));
		hipMemcpy(d_n, &(t_learnRate), sizeof(double), hipMemcpyHostToDevice);
		hipMalloc( (void **) &d_b, sizeof(double));
		hipMemcpy(d_b, &(t_parameterB), sizeof(double), hipMemcpyHostToDevice);

		hipMalloc( (void **) &d_inputSize, sizeof(int));
		hipMemcpy(d_inputSize, &(t_prevLayerReference.size), sizeof(int), hipMemcpyHostToDevice);
		inputSize = t_prevLayerReference.size;

		hipMalloc( (void **) &d_output, sizeof(double)*size);
		output = (double*) std::malloc(sizeof(double)*size);

		hipMalloc( (void **) &d_sums, sizeof(double)*size);

		hipMalloc( (void **) &d_weights, sizeof(double)*size*(inputSize+1));
		initWeights();

		hipMalloc( (void **) &d_deltas, sizeof(double)*size);
		deltas = (double*) malloc(sizeof(double)*size);
		de_prevDeltas = t_prevLayerReference.deltaPtr;

		learnRate = t_learnRate;

		numberOfBlocks = 1;
		while(1)
		{
			numberOfThreads = size/numberOfBlocks;
			if(numberOfThreads<=800 && numberOfThreads*numberOfBlocks==size) break;
			numberOfBlocks++;

			assert(numberOfBlocks < 10 && "Could not match thread/block size");
		}

	}

	/*
	 *
	 */
	SigmoidLayer::~SigmoidLayer()
	{
		hipFree(d_n);
		hipFree(d_b);

		hipFree(d_inputSize);
		hipFree(d_output);
		hipFree(d_sums);
		hipFree(d_weights);

		hipFree(d_deltas);

		free(output);
		free(deltas);
	}

	/*
	 *
	 */
	void SigmoidLayer::initWeights()
	{
		double *randomValues = (double*) malloc(sizeof(double)*size*(inputSize+1));

		for(int i=0; i<(inputSize+1)*size; i++)
		{
//			std::cout << (int) (100*i/((inputSize+1)*size)) << "%\n";
			double randomValue = getRandomWeight();
			randomValues[i] = randomValue;

		}
		hipMemcpy(d_weights, randomValues, sizeof(double)*size*(inputSize+1), hipMemcpyHostToDevice);
		free(randomValues);
	}

	/*
	 *
	 */
	std::vector<double> SigmoidLayer::getOutput()
	{
		hipMemcpy(output, d_output, sizeof(double)*size, hipMemcpyDeviceToHost);

		std::vector<double> result;
		for(int i=0; i<size; i++ )
		{
			double v = output[i];
			result.push_back(v);
		}

		return result;
	}

	void SigmoidLayer::determineOutput()
	{
		determineOutputFunc<<< numberOfThreads , numberOfBlocks >>>(de_input, d_output, d_inputSize, d_sums, d_weights, d_deltas, d_b);
	}

	void SigmoidLayer::setDelta(std::vector<double> t_z)
	{
		assert(t_z.size() == size && "learning values size not match");

		#pragma omp parallel for shared(deltas,size,output, t_z) private(i) default(none)
		for(int i=0; i<size; i++ )
		{
			deltas[i] = (double) output[i] - t_z[i];
		}

		hipMemcpy(d_deltas, deltas, sizeof(double)*size, hipMemcpyHostToDevice);
	}

	void SigmoidLayer::learnBackPropagation()
	{
//		int64 timeBefore = cv::getTickCount();
		learnBackPropagationFunc<<< numberOfThreads , numberOfBlocks >>>(de_input, d_inputSize, d_output, d_sums, d_weights, d_deltas, de_prevDeltas, d_n, d_b);
//		int64 afterBefore = cv::getTickCount();
//		std::cout << "Sigm: " << (afterBefore - timeBefore)/ cv::getTickFrequency() << "\n";
	}

	/*
	 *
	 */
	NeuronsPtr SigmoidLayer::getNeuronPtr()
	{
		return NeuronsPtr(d_output,size, d_deltas);
	}

	/*
	 *
	 */
//	void SigmoidLayer::saveToFile(std::ofstream & t_file)
//	{
//		t_file << (double) 1 << ' '; //Signature of SigmoidLayer
//		t_file << (double) neurons.size() << ' ';
//		t_file << learnRate << ' ';
//		t_file << b << ' ';
//
//		for( auto it = neurons.begin(); it != neurons.end(); it++)
//		{
//			std::vector<double> *weights = it->getWeights();
//			for(int i=0; i<weights->size(); i++)
//			{
//				t_file << (*weights)[i] << ' ';
//			}
//		}
//	}

	/*
	 *
	 */
//	void SigmoidLayer::loadFromFile(std::ifstream & t_file)
//	{
//		for( auto it = neurons.begin(); it != neurons.end(); it++)
//		{
//			double buff;
//			std::vector<double> *weights = it->getWeights();
//			for(int i=0; i<weights->size(); i++)
//			{
//				if(t_file.eof()) {assert("SigmoidLayer LoadFromFile: unexpected end of file");}
//				t_file >> buff;
//				(*weights)[i] = buff;
//			}
//		}
//	}
}
